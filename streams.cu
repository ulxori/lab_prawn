#include "hip/hip_runtime.h"
/*
CUDA - generation of array of N elements and calculates even and odd numbers occurence - with streams
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG 0
#define MAX 10
__host__ void errorexit(const char *s) {
    printf("\n%s\n",s); 
    exit(EXIT_FAILURE);   
}

__host__ void generate(int *matrix, int matrixSize) {
  srand(time(NULL));
  for(int i=0; i<matrixSize; i++) {
    matrix[i] = rand()%MAX;
  }
}

__global__ void calculation(int *matrix, int* histogram, int matrixSize, int streamChunk, int streamId) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x+streamId*streamChunk;

    if(my_index < matrixSize) {
      if(my_index < matrixSize) {
			  atomicAdd(&histogram[matrix[my_index]], 1);
		  } 
    } 
}

int main(int argc,char **argv) {

  ///define number of streams
  int numberOfStreams = 4;
  
  //define array size and allocate memory on host
  int matrixSize=12;
  int *hMatrix=(int*)malloc(matrixSize*sizeof(int));

  //get number of chunks to operate per stream
  int streamChunk = matrixSize/numberOfStreams;

  printf("Stream chunk is %d \n", streamChunk);
 
  //define kernel size per stream
  int threadsinblock=1000;
  int blocksingrid=1+((streamChunk-1)/threadsinblock); 

  printf("blocksingrid is %d \n", blocksingrid);
  
  //allocate memory for histogram - host
	int *hHistogram=(int*)malloc(MAX * sizeof(int));


  //create streams
  hipStream_t streams[numberOfStreams];
  for(int i=0;i<numberOfStreams;i++) {
      if (hipSuccess!=hipStreamCreate(&streams[i]))
           errorexit("Error creating stream");
    }

  if (hipSuccess!=hipHostMalloc((void **) &hMatrix, matrixSize*sizeof(int)))
      errorexit("Error allocating memory on the CPU");

  //generate random numbers
  generate(hMatrix, matrixSize);

  if(DEBUG) {
    printf("Generated numbers: \n");
    for(int i=0; i<matrixSize; i++) {
      printf("%d ", hMatrix[i]);
    }
    printf("\n");
  }

  //allocate memory for histogram and matrix- device
	int *dHistogram=NULL;
	int *dMatrix=NULL;

	if (hipSuccess!=hipMalloc((void **)&dMatrix,matrixSize*sizeof(int)))
			errorexit("Error allocating memory on the GPU");

	if (hipSuccess!=hipMalloc((void **)&dHistogram,MAX*sizeof(int)))
			errorexit("Error allocating memory on the GPU");

  //execute operation in each stream - copy chunk of data and run calculations
  for(int i=0; i<numberOfStreams; i++) {
    hipMemcpyAsync(&dMatrix[streamChunk*i],&hMatrix[streamChunk*i],streamChunk*sizeof(int),hipMemcpyHostToDevice, streams[i]);      
    calculation<<<blocksingrid, threadsinblock, threadsinblock*sizeof(double), streams[i]>>>(dMatrix, dHistogram, matrixSize, streamChunk, i);
  }

  hipDeviceSynchronize();

  //copy results from GPU
  if (hipSuccess!=hipMemcpy(hHistogram, dHistogram, MAX*sizeof(int),hipMemcpyDeviceToHost))
		 errorexit("Error copying results");

	for (int i=0; i<MAX; i++) {
		printf("[%d]=%d\n", i, hHistogram[i]);
	}

//Free memory and destroy streams
    for(int i=0;i<numberOfStreams;i++) {
      if (hipSuccess!=hipStreamDestroy(*(streams+i)))
         errorexit("Error creating stream");
    }

  free(hHistogram);
  
  if (hipSuccess!=hipHostFree(hMatrix))
     errorexit("Error when deallocating space on the CPU");
  if (hipSuccess!=hipFree(dHistogram))
    errorexit("Error when deallocating space on the GPU");
  if (hipSuccess!=hipFree(dMatrix))
    errorexit("Error when deallocating space on the GPU");
  
}
